// test.cu

#include "../include/cryptography.h"
#include "../include/definitions.h"
#include "../include/easylogging.h"
#include "../include/h0552230402key.h"
#include "../include/pre4867144607Hazh.h"
#include "../include/reduction.h"
#include "../include/request.h"
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <curl/curl.h>
#include <inttypes.h>
#include <iostream>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <atomic>
#include <chrono>
#include <mutex>
#include <thread>

INITIALIZE_EASYLOGGINGPP

namespace ch = std::chrono;

int TestSolutions(
    const info_t * info,
    const uint8_t * x,
    const uint8_t * w
)
{
    ctx_t ctx_h;

    uint32_t * bound_d;
    CUDA_CALL(hipMalloc(&bound_d, NUM_SIZE_8 + DATA_SIZE_8));
    uint32_t * data_d = bound_d + NUM_SIZE_32;

    uint32_t * hashes_d;
    CUDA_CALL(hipMalloc(&hashes_d, (uint32_t)N_LEN * NUM_SIZE_8));

    uint32_t * res_d;
    CUDA_CALL(hipMalloc(&res_d, WORKSPACE_SIZE_8));
    uint32_t * indices_d = res_d + NONCES_PER_ITER * NUM_SIZE_32;

    uctx_t * uctxs_d = NULL;

    if (info->keepPrehash)
    {
        CUDA_CALL(hipMalloc(&uctxs_d, (uint32_t)N_LEN * sizeof(uctx_t)));
    }

    CUDA_CALL(hipMemcpy(
        bound_d, info->bound, NUM_SIZE_8, hipMemcpyHostToDevice
    ));

    CUDA_CALL(hipMemcpy(data_d, info->pk, PK_SIZE_8, hipMemcpyHostToDevice));

    CUDA_CALL(hipMemcpy(
        (uint8_t *)data_d + PK_SIZE_8, info->mes, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    CUDA_CALL(hipMemcpy(
        (uint8_t *)data_d + PK_SIZE_8 + NUM_SIZE_8, w, PK_SIZE_8,
        hipMemcpyHostToDevice
    ));

    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + NUM_SIZE_32, x, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + 2 * NUM_SIZE_32, info->sk, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    uint64_t base = 0;

    if (info->keepPrehash)
    {
    }

    pre4867144607Hazh(info->keepPrehash, data_d, uctxs_d, hashes_d, res_d);
    CUDA_CALL(hipDeviceSynchronize());

    Inith0552230402key(&ctx_h, (uint32_t *)info->mes, NUM_SIZE_8);

    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + 3 * NUM_SIZE_32, &ctx_h, sizeof(ctx_t),
        hipMemcpyHostToDevice
    ));

    Blockh0552230402key<<<1 + (THREADS_PER_ITER - 1) / BLOCK_DIM, BLOCK_DIM>>>(
        bound_d, data_d, base, hashes_d, res_d, indices_d
    );

    uint64_t res_h[NUM_SIZE_64];
    uint32_t solFound = 0;
    uint32_t nonce;
    CUDA_CALL(hipMemcpy(
        res_h, res_d, NUM_SIZE_8,
        hipMemcpyDeviceToHost
    ));
    CUDA_CALL(hipMemcpy(
        &nonce, indices_d, sizeof(uint32_t),
        hipMemcpyDeviceToHost
    ));
    LOG(INFO) << "FNnce: " << nonce-1;
    if(nonce != 0x3381BF)
    {
        exit(EXIT_FAILURE);
    }

    CUDA_CALL(hipFree(bound_d));
    CUDA_CALL(hipFree(hashes_d));
    CUDA_CALL(hipFree(res_d));

    if (info->keepPrehash) { CUDA_CALL(hipFree(uctxs_d)); }

    return EXIT_SUCCESS;
}

int TestPerformance(
    const info_t * info,
    const uint8_t * x,
    const uint8_t * w
)
{

    ctx_t ctx_h;

    uint32_t * bound_d;
    CUDA_CALL(hipMalloc(&bound_d, NUM_SIZE_8 + DATA_SIZE_8));
    uint32_t * data_d = bound_d + NUM_SIZE_32;

    uint32_t * hashes_d;
    CUDA_CALL(hipMalloc(&hashes_d, (uint32_t)N_LEN * NUM_SIZE_8));

    uint32_t * res_d;
    CUDA_CALL(hipMalloc(&res_d, WORKSPACE_SIZE_8));
    uint32_t * indices_d = res_d + NONCES_PER_ITER * NUM_SIZE_32;

    uctx_t * uctxs_d = NULL;

    if (info->keepPrehash)
    {
        CUDA_CALL(hipMalloc(&uctxs_d, (uint32_t)N_LEN * sizeof(uctx_t)));
    }

    CUDA_CALL(hipMemcpy(
        bound_d, info->bound, NUM_SIZE_8, hipMemcpyHostToDevice
    ));

    CUDA_CALL(hipMemcpy(data_d, info->pk, PK_SIZE_8, hipMemcpyHostToDevice));

    CUDA_CALL(hipMemcpy(
        (uint8_t *)data_d + PK_SIZE_8, info->mes, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    CUDA_CALL(hipMemcpy(
        (uint8_t *)data_d + PK_SIZE_8 + NUM_SIZE_8, w, PK_SIZE_8,
        hipMemcpyHostToDevice
    ));

    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + NUM_SIZE_32, x, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + 2 * NUM_SIZE_32, info->sk, NUM_SIZE_8,
        hipMemcpyHostToDevice
    ));

    uint64_t base = 0;

    ch::milliseconds ms = ch::milliseconds::zero(); 

    ch::milliseconds start = ch::duration_cast<ch::milliseconds>(
        ch::system_clock::now().time_since_epoch()
    );

    Prehash(0, data_d, NULL, hashes_d, res_d);

    CUDA_CALL(hipDeviceSynchronize());
    
    ms = ch::duration_cast<ch::milliseconds>(
        ch::system_clock::now().time_since_epoch()
    ) - start;

    LOG(INFO) << "Pre Time: " << ms.count() << " ms";

    if (info->keepPrehash)
    {

        CUDA_CALL(hipDeviceSynchronize());

        start = ch::duration_cast<ch::milliseconds>(
            ch::system_clock::now().time_since_epoch()
        );

        Prehash(1, data_d, uctxs_d, hashes_d, res_d);

        CUDA_CALL(hipDeviceSynchronize());

        ms = ch::duration_cast<ch::milliseconds>(
            ch::system_clock::now().time_since_epoch()
        ) - start;

    }

    CUDA_CALL(hipDeviceSynchronize());

    InitMini(&ctx_h, (uint32_t *)info->mes, NUM_SIZE_8);

    CUDA_CALL(hipMemcpy(
        data_d + COUPLED_PK_SIZE_32 + 3 * NUM_SIZE_32, &ctx_h, sizeof(ctx_t),
        hipMemcpyHostToDevice
    ));

    ms = ch::milliseconds::zero();

    uint32_t sum = 0;
    int iter = 0;
    uint32_t nonce = 0;
    start = ch::duration_cast<ch::milliseconds>(
        ch::system_clock::now().time_since_epoch()
    );

    for ( ; ms.count() < 60000; ++iter)
    {
        BlockHkey<<<1 + (THREADS_PER_ITER - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            bound_d, data_d, base, hashes_d, res_d, indices_d
        );

        CUDA_CALL(hipMemcpy(
            &nonce, indices_d, sizeof(uint32_t),
            hipMemcpyDeviceToHost
        ));

        if(nonce != 0) ++sum;

        CUDA_CALL(hipMemset(indices_d, 0 ,sizeof(uint32_t)));
        hipDeviceSynchronize();
        base += NONCES_PER_ITER;

        ms = ch::duration_cast<ch::milliseconds>(
            ch::system_clock::now().time_since_epoch()
        ) - start;
    }

    CUDA_CALL(hipFree(bound_d));
    CUDA_CALL(hipFree(hashes_d));
    CUDA_CALL(hipFree(res_d));

    if (info->keepPrehash) { CUDA_CALL(hipFree(uctxs_d)); }

    return EXIT_SUCCESS;
}


void TestRequests()
{
    json_t oldreq(0, REQ_LEN);
    json_t *newreq;
    newreq = new json_t(0, REQ_LEN);
    json_t oldreqbig(0, REQ_LEN);
    info_t testinfo;

    char bigrequest[] = "{ \"msg\" : \"46b7e94915275125129581725817295812759128"
                        "571925871285728572857285725285728571928517287519285718"
                        "275192857192857192857192587129581729587129581728571295"
                        "817295182759128751928571925871285782758782751928571827"
                        "519285787bfad202ab4e3dd9cc0603c1f61f53485854028b8fa03f"
                        "399544fb298\", \"b\" : 2134827235332678044033321050158"
                        "7889707005372997724693988999057291299,  \"pk\" : \"039"
                        "5f8d54fdd5edb7eeab3228c952d39f5e60d048178f94ac992d4f76"
                        "a6dce4c71\"  }";
    WriteFunc((void*)bigrequest, sizeof(char), strlen(bigrequest), &oldreqbig);
    if(strcmp(bigrequest, oldreqbig.ptr))
    {
    }
    
    
    char request[] = "{ \"msg\" : \"46b7e949bfad202ab4e3dd9cc0603c1f61f5348585"
                     "4028b8fa03f399544fb298\", \"b\" : 2134827235332678044033"
                     "3210501587889707005372997724693988999057291299,  \"pk\" "
                     ": \"0395f8d54fdd5edb7eeab3228c952d39f5e60d048178f94ac992"
                     "d4f76a6dce4c71\"  }";

    WriteFunc((void*)request, sizeof(char), strlen(request), &oldreq);
    if(strcmp(request, oldreq.ptr))
    {
    }
    


    char seedstring[] = "13cc81ef0b13fd496217c7c44b16c09d923ad475d897cffd37c63"
                        "a15aebf579313d67934727d94ba42687f238480eb9248da9ba21e9c1";

    GenerateSecKey(
        seedstring, strlen(seedstring), testinfo.sk,
        testinfo.skstr
    );
    GeneratePublicKey(testinfo.skstr, testinfo.pkstr, testinfo.pk);

    char shortrequest[] =  "{ \"msg\" : \"46b7e\", \"b\" : 2134,  \"pk\" : \"0395"
                            "f8d54fdd5edb7eeab3228c952d39f5e60d048178f94ac992d4"
                            "f76a6dce4c71\"  }";
    char brokenrequest[] =  " \"msg\"  \"46b7e\", \"b\" : 2134,  \"pk\" : \"0395f8"
                            "d54fdd5edb7eeab3228c952d39f5e60d048178f94ac992d4f76a6"
                            "dce4c71\"  }";
    char uncompleterequest[] =  "{ \"msg\" : \"46b7e\", \"pk\" : \"0395f8d54fdd5edb"
                                "7eeab3228c952d39f5e60d048178f94ac992d4f76a6dce4c71\""
                                " }";
    char uncompleterequest2[] =  "{ \"b\" : 2134,  \"pk\" : \"0395f8d54fdd5edb7eeab"
                                 "3228c952d39f5e60d048178f94ac992d4f76a6dce4c71\"  }";

    WriteFunc((void*)shortrequest, sizeof(char), strlen(shortrequest), newreq);
    LOG(INFO) << "Testing short request "
     << "\n result " << ((ParseRequest(&oldreq, newreq, &testinfo, 1) == EXIT_SUCCESS) ? "OK" : "ERROR");
    delete newreq;
    newreq = new json_t(0, REQ_LEN);
    WriteFunc((void*)bigrequest, sizeof(char), strlen(bigrequest), newreq);
     LOG(INFO) << "Testing big request " 
      << "\n result " << ((ParseRequest(&oldreq, newreq, &testinfo, 1) == EXIT_SUCCESS) ? "OK" : "ERROR");
    delete newreq;
    newreq = new json_t(0, REQ_LEN);
    WriteFunc((void*)brokenrequest, sizeof(char), strlen(brokenrequest), newreq);
      LOG(INFO) << "Testing broken request " 
       << "\n result " << ((ParseRequest(&oldreq, newreq, &testinfo, 1) == EXIT_SUCCESS) ? "ERROR" : "OK");
    delete newreq;
    newreq = new json_t(0, REQ_LEN);
    WriteFunc((void*)uncompleterequest, sizeof(char), strlen(uncompleterequest), newreq);
       LOG(INFO) << "Testing uncomplete request 1 " 
        << "\n result " << ((ParseRequest(&oldreq, newreq, &testinfo, 1) == EXIT_SUCCESS) ? "ERROR" : "OK");
    delete newreq;
    newreq = new json_t(0, REQ_LEN);
    WriteFunc((void*)uncompleterequest2, sizeof(char), strlen(uncompleterequest2), newreq);
    LOG(INFO) << "Testing uncomplete request 2 " 
     << "\n result " << ((ParseRequest(&oldreq, newreq, &testinfo, 1) == EXIT_SUCCESS) ? "ERROR" : "OK");
    delete newreq;
}


void TestNewCrypt()
{
    char mnemonic[] = "edge talent poet tortoise trumpet dose";
    uint8_t sk[NUM_SIZE_8];
    char skstr[NUM_SIZE_4];
    char pkstr[PK_SIZE_4+1];
    uint8_t pk[PK_SIZE_8];

    GenerateSecKeyNew(mnemonic, strlen(mnemonic), sk, skstr, "");
    
    if(strncmp(skstr, "392F75AD23278B3CD7B060D900138F20F8CBA89ABB259B5DCF5D9830B49D8E38", NUM_SIZE_4))
    {
        printf("%.64s private key1\n", skstr);
    }
    else
    {
    }
}


int main(int argc, char ** argv)
{
    START_EASYLOGGINGPP(argc, argv);

    el::Loggers::reconfigureAllLoggers(
        el::ConfigurationType::Format, "%datetime %level [%thread] %msg"
    );

    el::Helpers::setThreadName("test thread");

    TestNewCrypt();

    TestRequests();
    int deviceCount;

    if (hipGetDeviceCount(&deviceCount) != hipSuccess)
    {
        exit(EXIT_FAILURE);
    }

    size_t freeMem;
    size_t totalMem;

    CUDA_CALL(hipMemGetInfo(&freeMem, &totalMem));
    
    if (freeMem < MIN_FREE_MEMORY)
    {
        exit(EXIT_FAILURE);
    }
    
    info_t info;
    uint8_t x[NUM_SIZE_8];
    uint8_t w[PK_SIZE_8];
    char seed[256] = "Va'esse deireadh aep eigean, va'esse eigh faidh'ar";

    GenerateSecKey(seed, 50, info.sk, info.skstr);
    GeneratePublicKey(info.skstr, info.pkstr, info.pk);

    const char ref_pkstr[PK_SIZE_4 + 1]
        = "020C16DFC5E23C59357E89D44977038F0A7851CC9926B3AABB3FF9E7E6A57315AD";

    int test = !strncmp(ref_pkstr, info.pkstr, PK_SIZE_4);

    if (!test)
    {
        return EXIT_FAILURE;
    }

    ((uint64_t *)info.bound)[0] = 0xFFFFFFFFFFFFFFFF;
    ((uint64_t *)info.bound)[1] = 0xFFFFFFFFFFFFFFFF;
    ((uint64_t *)info.bound)[2] = 0xFFFFFFFFFFFFFFFF;
    ((uint64_t *)info.bound)[3] = 0x000002FFFFFFFFFF;

    ((uint64_t *)info.mes)[0] = 1;
    ((uint64_t *)info.mes)[1] = 0;
    ((uint64_t *)info.mes)[2] = 0;
    ((uint64_t *)info.mes)[3] = 0;

    sprintf(seed, "%d", 0);

    GenerateSecKey(seed, 1, x, info.skstr);
    GeneratePublicKey(info.skstr, info.pkstr, w);

    if (NONCES_PER_ITER <= 0x3D5B84)
    {
    }
    else
    {
        info.keepPrehash = 0;
        TestSolutions(&info, x, w);

        if (freeMem < MIN_FREE_MEMORY_PREHASH)
        {
        }
        else
        {
            info.keepPrehash = 1;
            TestSolutions(&info, x, w);
        }
    }

    info.keepPrehash = (freeMem >= MIN_FREE_MEMORY_PREHASH)? 1: 0;
    TestPerformance(&info, x, w);

    return EXIT_SUCCESS;
}

// test.cu
