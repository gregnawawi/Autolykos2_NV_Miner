#include "hip/hip_runtime.h"
// h0552230402key.cu

#include "../include/h0552230402key.h"
#include <hip/hip_runtime.h>

__device__ __forceinline__ uint32_t ld_gbl_cs(const  uint32_t * __restrict__ p) {
  uint32_t v;
  asm("ld.global.cs.u32 %0, [%1];" : "=r"(v) : "l"(p));
  return v;
}

__device__ __forceinline__ uint4 ld_gbl_cs_v4(const  uint4 * __restrict__ p) {
  uint4 v;
  asm("ld.global.cs.v4.u32 {%0, %1, %2, %3}, [%4];" : "=r"(v.x), "=r"(v.y), "=r"(v.z), "=r"(v.w) : "l"(p));
  return v;
}

__device__ __forceinline__ uint32_t cuda_swab32(uint32_t x)
{
  return __byte_perm(x, x, 0x0123);
}

__device__ __forceinline__ uint64_t devectorize(uint2 x)
{
  uint64_t result;
  asm("mov.b64 %0,{%1,%2}; \n\t"
    : "=l"(result) : "r"(x.x), "r"(x.y));
  return result;
}


__device__ __forceinline__ uint2 vectorize(const uint64_t x)
{
  uint2 result;
  asm("mov.b64 {%0,%1},%2; \n\t"
    : "=r"(result.x), "=r"(result.y) : "l"(x));
  return result;
}

__device__ __forceinline__
uint64_t devROTR64(uint64_t b, int offset)
{
  uint2 a;
  uint2 result;
  a = vectorize(b);

  if (offset < 32) {
    asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
    asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
  }
  else {
    asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
    asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
  }
  return devectorize(result);
}

__device__ __forceinline__
uint2 __byte_perm_64(const uint2 source, const uint32_t grab1, const uint32_t grab2)
{
  uint2 r;
  asm("prmt.b32 %0, %1, %2, %3;" : "=r"(r.x) : "r"(source.x), "r"(source.y), "r"(grab1));
  asm("prmt.b32 %0, %1, %2, %3;" : "=r"(r.y) : "r"(source.x), "r"(source.y), "r"(grab2));
  return r;
}

__device__ __forceinline__
uint2 __swap_hilo(const uint2 source)
{
  uint2 r;

  r.x = source.y;
  r.y = source.x;

  return r;
}

__device__ __forceinline__
void devB2B_G(uint64_t* v, int a, int b, int c, int d, uint64_t x, uint64_t y)
{
    ((uint64_t *)(v))[a] += ((uint64_t *)(v))[b] + x;
    ((uint64_t *)(v))[d]
        = devROTR64(((uint64_t *)(v))[d] ^ ((uint64_t *)(v))[a], 32);
    ((uint64_t *)(v))[c] += ((uint64_t *)(v))[d];
    ((uint64_t *)(v))[b]
        = devROTR64(((uint64_t *)(v))[b] ^ ((uint64_t *)(v))[c], 24);
    ((uint64_t *)(v))[a] += ((uint64_t *)(v))[b] + y;
    ((uint64_t *)(v))[d]
        = devROTR64(((uint64_t *)(v))[d] ^ ((uint64_t *)(v))[a], 16);
    ((uint64_t *)(v))[c] += ((uint64_t *)(v))[d];
    ((uint64_t *)(v))[b]
        = devROTR64(((uint64_t *)(v))[b] ^ ((uint64_t *)(v))[c], 63);
}


__device__ __forceinline__
void devB2B_MIX(uint64_t* v, uint64_t* m)                                                                                                                                 \
{
  devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[0], ((uint64_t *)(m))[1]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[2], ((uint64_t *)(m))[3]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[4], ((uint64_t *)(m))[5]);
  devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[6], ((uint64_t *)(m))[7]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[8], ((uint64_t *)(m))[9]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[10], ((uint64_t *)(m))[11]);
    devB2B_G(v, 2, 7,  8, 13, ((uint64_t *)(m))[12], ((uint64_t *)(m))[13]);
    devB2B_G(v, 3, 4,  9, 14, ((uint64_t *)(m))[14], ((uint64_t *)(m))[15]);

    devB2B_G(v, 0, 4,  8, 12, ((uint64_t *)(m))[14], ((uint64_t *)(m))[10]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[4], ((uint64_t *)(m))[8]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[9], ((uint64_t *)(m))[15]);
  devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[13], ((uint64_t *)(m))[6]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[1], ((uint64_t *)(m))[12]);
  devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[0], ((uint64_t *)(m))[2]);
  devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[11], ((uint64_t *)(m))[7]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[5], ((uint64_t *)(m))[3]);

  devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[11], ((uint64_t *)(m))[8]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[12], ((uint64_t *)(m))[0]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[5], ((uint64_t *)(m))[2]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[15], ((uint64_t *)(m))[13]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[10], ((uint64_t *)(m))[14]);
  devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[3], ((uint64_t *)(m))[6]);
  devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[7], ((uint64_t *)(m))[1]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[9], ((uint64_t *)(m))[4]);

  devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[7], ((uint64_t *)(m))[9]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[3], ((uint64_t *)(m))[1]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[13], ((uint64_t *)(m))[12]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[11], ((uint64_t *)(m))[14]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[2], ((uint64_t *)(m))[6]);
  devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[5], ((uint64_t *)(m))[10]);
  devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[4], ((uint64_t *)(m))[0]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[15], ((uint64_t *)(m))[8]);

  devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[9], ((uint64_t *)(m))[0]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[5], ((uint64_t *)(m))[7]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[2], ((uint64_t *)(m))[4]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[10], ((uint64_t *)(m))[15]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[14], ((uint64_t *)(m))[1]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[11], ((uint64_t *)(m))[12]);
  devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[6], ((uint64_t *)(m))[8]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[3], ((uint64_t *)(m))[13]);

  devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[2], ((uint64_t *)(m))[12]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[6], ((uint64_t *)(m))[10]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[0], ((uint64_t *)(m))[11]);
  devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[8], ((uint64_t *)(m))[3]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[4], ((uint64_t *)(m))[13]);
  devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[7], ((uint64_t *)(m))[5]);
    devB2B_G(v, 2, 7,  8, 13, ((uint64_t *)(m))[15], ((uint64_t *)(m))[14]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[1], ((uint64_t *)(m))[9]);

  devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[12], ((uint64_t *)(m))[5]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[1], ((uint64_t *)(m))[15]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[14], ((uint64_t *)(m))[13]);
  devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[4], ((uint64_t *)(m))[10]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[0], ((uint64_t *)(m))[7]);
  devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[6], ((uint64_t *)(m))[3]);
  devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[9], ((uint64_t *)(m))[2]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[8], ((uint64_t *)(m))[11]);

    devB2B_G(v, 0, 4,  8, 12, ((uint64_t *)(m))[13], ((uint64_t *)(m))[11]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[7], ((uint64_t *)(m))[14]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[12], ((uint64_t *)(m))[1]);
  devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[3], ((uint64_t *)(m))[9]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[5], ((uint64_t *)(m))[0]);
  devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[15], ((uint64_t *)(m))[4]);
  devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[8], ((uint64_t *)(m))[6]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[2], ((uint64_t *)(m))[10]);

  devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[6], ((uint64_t *)(m))[15]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[14], ((uint64_t *)(m))[9]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[11], ((uint64_t *)(m))[3]);
  devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[0], ((uint64_t *)(m))[8]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[12], ((uint64_t *)(m))[2]);
  devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[13], ((uint64_t *)(m))[7]);
  devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[1], ((uint64_t *)(m))[4]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[10], ((uint64_t *)(m))[5]);

  devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[10], ((uint64_t *)(m))[2]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[8], ((uint64_t *)(m))[4]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[7], ((uint64_t *)(m))[6]);
  devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[1], ((uint64_t *)(m))[5]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[15], ((uint64_t *)(m))[11]);
  devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[9], ((uint64_t *)(m))[14]);
  devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[3], ((uint64_t *)(m))[12]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[13], ((uint64_t *)(m))[0]);

  devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[0], ((uint64_t *)(m))[1]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[2], ((uint64_t *)(m))[3]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[4], ((uint64_t *)(m))[5]);
  devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[6], ((uint64_t *)(m))[7]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[8], ((uint64_t *)(m))[9]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[10], ((uint64_t *)(m))[11]);
    devB2B_G(v, 2, 7,  8, 13, ((uint64_t *)(m))[12], ((uint64_t *)(m))[13]);
    devB2B_G(v, 3, 4,  9, 14, ((uint64_t *)(m))[14], ((uint64_t *)(m))[15]);

    devB2B_G(v, 0, 4,  8, 12, ((uint64_t *)(m))[14], ((uint64_t *)(m))[10]);
  devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[4], ((uint64_t *)(m))[8]);
  devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[9], ((uint64_t *)(m))[15]);
  devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[13], ((uint64_t *)(m))[6]);
  devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[1], ((uint64_t *)(m))[12]);
  devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[0], ((uint64_t *)(m))[2]);
  devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[11], ((uint64_t *)(m))[7]);
  devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[5], ((uint64_t *)(m))[3]);
}

__device__ __forceinline__
void devDEVICE_B2B_H_LAST(ctx_t *ctx, uint64_t* aux)                                                                                                                   \
{
    asm volatile (
        "add.cc.u32 %0, %0, %1;":
  "+r"(((uint32_t *)((ctx_t *)(ctx))->t)[0]) :
        "r"(((ctx_t *)(ctx))->c)
    );
    asm volatile (
        "addc.cc.u32 %0, %0, 0;":
        "+r"(((uint32_t *)((ctx_t *)(ctx))->t)[1])
    );
    asm volatile (
        "addc.cc.u32 %0, %0, 0;":
        "+r"(((uint32_t *)((ctx_t *)(ctx))->t)[2])
    );
    asm volatile (
        "addc.u32 %0, %0, 0;":
        "+r"(((uint32_t *)((ctx_t *)(ctx))->t)[3])
    );

    while (((ctx_t *)(ctx))->c < BUF_SIZE_8)
    {
        ((ctx_t *)(ctx))->b[((ctx_t *)(ctx))->c++] = 0;
    }

    ((uint64_t *)(aux))[0] = ((ctx_t *)(ctx))->h[0];
    ((uint64_t *)(aux))[1] = ((ctx_t *)(ctx))->h[1];
    ((uint64_t *)(aux))[2] = ((ctx_t *)(ctx))->h[2];
    ((uint64_t *)(aux))[3] = ((ctx_t *)(ctx))->h[3];
    ((uint64_t *)(aux))[4] = ((ctx_t *)(ctx))->h[4];
    ((uint64_t *)(aux))[5] = ((ctx_t *)(ctx))->h[5];
    ((uint64_t *)(aux))[6] = ((ctx_t *)(ctx))->h[6];
    ((uint64_t *)(aux))[7] = ((ctx_t *)(ctx))->h[7];

    B2B_IV(aux + 8);

    ((uint64_t *)(aux))[12] ^= ((ctx_t *)(ctx))->t[0];
    ((uint64_t *)(aux))[13] ^= ((ctx_t *)(ctx))->t[1];

    ((uint64_t *)(aux))[14] = ~((uint64_t *)(aux))[14];

  ((uint64_t *)(aux))[16] = ((uint64_t *)(((ctx_t *)(ctx))->b))[0];
  ((uint64_t *)(aux))[17] = ((uint64_t *)(((ctx_t *)(ctx))->b))[1];
  ((uint64_t *)(aux))[18] = ((uint64_t *)(((ctx_t *)(ctx))->b))[2];
  ((uint64_t *)(aux))[19] = ((uint64_t *)(((ctx_t *)(ctx))->b))[3];
  ((uint64_t *)(aux))[20] = ((uint64_t *)(((ctx_t *)(ctx))->b))[4];
  ((uint64_t *)(aux))[21] = ((uint64_t *)(((ctx_t *)(ctx))->b))[5];
  ((uint64_t *)(aux))[22] = ((uint64_t *)(((ctx_t *)(ctx))->b))[6];
  ((uint64_t *)(aux))[23] = ((uint64_t *)(((ctx_t *)(ctx))->b))[7];
  ((uint64_t *)(aux))[24] = ((uint64_t *)(((ctx_t *)(ctx))->b))[8];
  ((uint64_t *)(aux))[25] = ((uint64_t *)(((ctx_t *)(ctx))->b))[9];
    ((uint64_t *)(aux))[26] = ((uint64_t *)(((ctx_t *)(ctx))->b))[10];
    ((uint64_t *)(aux))[27] = ((uint64_t *)(((ctx_t *)(ctx))->b))[11];
    ((uint64_t *)(aux))[28] = ((uint64_t *)(((ctx_t *)(ctx))->b))[12];
    ((uint64_t *)(aux))[29] = ((uint64_t *)(((ctx_t *)(ctx))->b))[13];
    ((uint64_t *)(aux))[30] = ((uint64_t *)(((ctx_t *)(ctx))->b))[14];
    ((uint64_t *)(aux))[31] = ((uint64_t *)(((ctx_t *)(ctx))->b))[15];

    devB2B_MIX(aux, aux + 16);

  ((ctx_t *)(ctx))->h[0] ^= ((uint64_t *)(aux))[0] ^ ((uint64_t *)(aux))[8];
  ((ctx_t *)(ctx))->h[1] ^= ((uint64_t *)(aux))[1] ^ ((uint64_t *)(aux))[9];
    ((ctx_t *)(ctx))->h[2] ^= ((uint64_t *)(aux))[2] ^ ((uint64_t *)(aux))[10];
    ((ctx_t *)(ctx))->h[3] ^= ((uint64_t *)(aux))[3] ^ ((uint64_t *)(aux))[11];
    ((ctx_t *)(ctx))->h[4] ^= ((uint64_t *)(aux))[4] ^ ((uint64_t *)(aux))[12];
    ((ctx_t *)(ctx))->h[5] ^= ((uint64_t *)(aux))[5] ^ ((uint64_t *)(aux))[13];
    ((ctx_t *)(ctx))->h[6] ^= ((uint64_t *)(aux))[6] ^ ((uint64_t *)(aux))[14];
    ((ctx_t *)(ctx))->h[7] ^= ((uint64_t *)(aux))[7] ^ ((uint64_t *)(aux))[15];

  return;
}

const __constant__ uint64_t ivals[8] = {
    0x6A09E667F2BDC928,
    0xBB67AE8584CAA73B,
    0x3C6EF372FE94F82B,
    0xA54FF53A5F1D36F1,
    0x510E527FADE682D1,
    0x9B05688C2B3E6C1F,
    0x1F83D9ABFB41BD6B,
    0x5BE0CD19137E2179
};

void cpyBSymbol(uint8_t *bound)
{
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(bound_), bound, NUM_SIZE_32 * sizeof(uint32_t)));
}

void Inith0552230402key(
    ctx_t * ctx,
    const uint32_t * mes,
    const uint32_t meslen
)
{
    
    memset(ctx->b, 0, BUF_SIZE_8);
    B2B_IV(ctx->h);
    ctx->h[0] ^= 0x01010000 ^ NUM_SIZE_8;
    memset(ctx->t, 0, 16);
    ctx->c = 0;

    for (uint_t j = 0; j < meslen; ++j)
    {
        ctx->b[ctx->c++] = ((const uint8_t *)mes)[j];
    }
    ((uint32_t*)(ctx->t))[0] = 40;
    ctx->c = 40;
    return;
}

__global__ __launch_bounds__(64, 64)
__global__ void Blockh0552230402keyStep1(


    const uint32_t * data,

    const uint64_t base,

    const uint32_t * hashes,

    uint32_t* BHashes
)

{

    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t r[9] = { 0 };

    uint64_t aux[32];

    uint32_t j;
    uint32_t non[NONCE_SIZE_32];
    uint64_t tmp;
    uint64_t hsh;
    uint64_t h2;
  uint32_t h3;

  for (int ii = 0; ii < 4; ii++)
    {
    tid = (NONCES_PER_ITER / 4)*ii + threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < NONCES_PER_ITER)
        {
        asm volatile (
            "add.cc.u32 %0, %1, %2;":
      "=r"(non[0]) : "r"(((uint32_t *)&base)[0]), "r"(tid)
        );

        asm volatile (
        "addc.u32 %0, %1, 0;": "=r"(non[1]) : "r"(((uint32_t *)&base)[1])
        );


      ((uint32_t*)(&tmp))[0] = __byte_perm(non[1], 0, 0x0123);
      ((uint32_t*)(&tmp))[1] = __byte_perm(non[0], 0, 0x0123);

        B2B_IV(aux);
        B2B_IV(aux + 8);
        aux[0] = ivals[0];
        ((uint64_t *)(aux))[12] ^= 40;
        ((uint64_t *)(aux))[13] ^= 0;

        ((uint64_t *)(aux))[14] = ~((uint64_t *)(aux))[14];

      ((uint64_t *)(aux))[16] = ((uint64_t *)data)[0];
      ((uint64_t *)(aux))[17] = ((uint64_t *)data)[1];
      ((uint64_t *)(aux))[18] = ((uint64_t *)data)[2];
      ((uint64_t *)(aux))[19] = ((uint64_t *)data)[3];
        ((uint64_t *)(aux))[20] = tmp;
        ((uint64_t *)(aux))[21] = 0;
      ((uint64_t *)(aux))[22] = 0;
      ((uint64_t *)(aux))[23] = 0;
      ((uint64_t *)(aux))[24] = 0;
      ((uint64_t *)(aux))[25] = 0;
      ((uint64_t *)(aux))[26] = 0;
      ((uint64_t *)(aux))[27] = 0;
      ((uint64_t *)(aux))[28] = 0;
      ((uint64_t *)(aux))[29] = 0;
      ((uint64_t *)(aux))[30] = 0;
      ((uint64_t *)(aux))[31] = 0;



        devB2B_MIX(aux, aux + 16);


#pragma unroll
      for (j = 0; j < NUM_SIZE_32; j += 2)
        {
            hsh = ivals[j >> 1];
        hsh ^= ((uint64_t *)(aux))[j >> 1] ^ ((uint64_t *)(aux))[8 + (j >> 1)];

            r[j] =  ((uint32_t*)(&hsh))[0];
        r[j + 1] = ((uint32_t*)(&hsh))[1];
        }

        ((uint8_t*)&h2)[0] = ((uint8_t*)r)[31];
        ((uint8_t*)&h2)[1] = ((uint8_t*)r)[30];
        ((uint8_t*)&h2)[2] = ((uint8_t*)r)[29];
        ((uint8_t*)&h2)[3] = ((uint8_t*)r)[28];
        ((uint8_t*)&h2)[4] = ((uint8_t*)r)[27];
        ((uint8_t*)&h2)[5] = ((uint8_t*)r)[26];
        ((uint8_t*)&h2)[6] = ((uint8_t*)r)[25];
        ((uint8_t*)&h2)[7] = ((uint8_t*)r)[24];

        h3 = h2 % N_LEN;

#pragma unroll 8
    for (int i = 0; i < 8; ++i)
    {
        r[7 - i] = cuda_swab32(hashes[(h3 << 3) + i]);
    }

             B2B_IV(aux);
             B2B_IV(aux + 8);
             aux[0] = ivals[0];
      ((uint64_t *)(aux))[12] ^= 71;//31+32+8;
             ((uint64_t *)(aux))[13] ^= 0;

             ((uint64_t *)(aux))[14] = ~((uint64_t *)(aux))[14];

      uint8_t *bb = (uint8_t *)(&(((uint64_t *)(aux))[16]));
      ((uint64_t *)bb)[0] = ((uint64_t *)(&((uint8_t *)r)[1]))[0];
      ((uint64_t *)bb)[1] = ((uint64_t *)(&((uint8_t *)r)[1]))[1];
      ((uint64_t *)bb)[2] = ((uint64_t *)(&((uint8_t *)r)[1]))[2];
      ((uint64_t *)bb)[3] = ((uint64_t *)(&((uint8_t *)r)[1]))[3];

      ((uint64_t *)&bb[31])[0] = ((uint64_t *)data)[0];
      ((uint64_t *)&bb[39])[0] = ((uint64_t *)data)[1];
      ((uint64_t *)&bb[47])[0] = ((uint64_t *)data)[2];
      ((uint64_t *)&bb[55])[0] = ((uint64_t *)data)[3];

      ((uint64_t *)&bb[63])[0] = tmp;

      ((uint64_t *)(aux))[25] = 0;
      ((uint64_t *)(aux))[26] = 0;
      ((uint64_t *)(aux))[27] = 0;
      ((uint64_t *)(aux))[28] = 0;
      ((uint64_t *)(aux))[29] = 0;
      ((uint64_t *)(aux))[30] = 0;
      ((uint64_t *)(aux))[31] = 0;

             devB2B_MIX(aux, aux + 16);

#pragma unroll
      for (j = 0; j < NUM_SIZE_32; j += 2)
             {
                 hsh = ivals[j >> 1];
        hsh ^= ((uint64_t *)(aux))[j >> 1] ^ ((uint64_t *)(aux))[8 + (j >> 1)];
        BHashes[THREADS_PER_ITER*j + tid] = __byte_perm(((uint32_t*)(&hsh))[0], 0, 0x0123);
        BHashes[THREADS_PER_ITER*(j + 1) + tid] = __byte_perm(((uint32_t*)(&hsh))[1], 0, 0x0123);
      }




        }
        }




    return;

}


__global__ __launch_bounds__(64, 64)
__global__ void Blockh0552230402keyStep2(
    const uint32_t * data,
    const uint64_t base,
    const uint32_t height,
    const uint32_t * hashes,
  uint32_t * valid,
    uint32_t * count,
    uint32_t*  BHashes
)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t const thread_id = threadIdx.x & 7;
  uint32_t const thrdblck_id = threadIdx.x;
  uint32_t const hash_id = threadIdx.x >> 3;

  uint64_t aux[32] = { 0 };
  uint32_t ind[32] = { 0 };
  uint32_t r[9] = { 0 };

  uint4 v1 = { 0,0,0,0 };
  uint4 v2 = { 0,0,0,0 };
  uint4 v3 = { 0,0,0,0 };
  uint4 v4 = { 0,0,0,0 };

    ctx_t sdata;
  ctx_t *ctx = ((ctx_t *)(&sdata));

  __shared__ uint32_t shared_index[64];
  __shared__ uint32_t shared_data[512];

    uint8_t j = 0;

    if (tid < NONCES_PER_ITER)
    {

#pragma unroll
    for (int k = 0; k < 8; k++)
        {
            r[k] = (BHashes[k*THREADS_PER_ITER + tid]);
        }


        ((uint8_t *)r)[32] = ((uint8_t *)r)[0];
    ((uint8_t *)r)[33] = ((uint8_t *)r)[1];
    ((uint8_t *)r)[34] = ((uint8_t *)r)[2];
    ((uint8_t *)r)[35] = ((uint8_t *)r)[3];

#pragma unroll
        for (int k = 0; k < K_LEN; k += 4)
        {
            ind[k] = r[k >> 2] & N_MASK;
            ind[k + 1] = ((r[k >> 2] << 8) | (r[(k >> 2) + 1] >> 24)) & N_MASK;
      ind[k + 2] = ((r[k >> 2] << 16) | (r[(k >> 2) + 1] >> 16)) & N_MASK;
      ind[k + 3] = ((r[k >> 2] << 24) | (r[(k >> 2) + 1] >> 8)) & N_MASK;
        }

    shared_index[thrdblck_id] = ind[0];
    __syncthreads();

    shared_data[(hash_id << 3) + thread_id] = (hashes[(shared_index[hash_id] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 64] = (hashes[(shared_index[hash_id + 8] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 128] = (hashes[(shared_index[hash_id + 16] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 192] = (hashes[(shared_index[hash_id + 24] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 256] = (hashes[(shared_index[hash_id + 32] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 320] = (hashes[(shared_index[hash_id + 40] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 384] = (hashes[(shared_index[hash_id + 48] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 448] = (hashes[(shared_index[hash_id + 56] << 3) + thread_id]);
    __syncthreads();

    v1.x = shared_data[(thrdblck_id << 3) + 0];
    v1.y = shared_data[(thrdblck_id << 3) + 1];
    v1.z = shared_data[(thrdblck_id << 3) + 2];
    v1.w = shared_data[(thrdblck_id << 3) + 3];
    v3.x = shared_data[(thrdblck_id << 3) + 4];
    v3.y = shared_data[(thrdblck_id << 3) + 5];
    v3.z = shared_data[(thrdblck_id << 3) + 6];
    v3.w = shared_data[(thrdblck_id << 3) + 7];

    shared_index[thrdblck_id] = ind[1];
    __syncthreads();

    shared_data[(hash_id << 3) + thread_id] = (hashes[(shared_index[hash_id] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 64] = (hashes[(shared_index[hash_id + 8] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 128] = (hashes[(shared_index[hash_id + 16] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 192] = (hashes[(shared_index[hash_id + 24] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 256] = (hashes[(shared_index[hash_id + 32] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 320] = (hashes[(shared_index[hash_id + 40] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 384] = (hashes[(shared_index[hash_id + 48] << 3) + thread_id]);
    shared_data[(hash_id << 3) + thread_id + 448] = (hashes[(shared_index[hash_id + 56] << 3) + thread_id]);
    __syncthreads();

    v2.x = shared_data[(thrdblck_id << 3) + 0];
    v2.y = shared_data[(thrdblck_id << 3) + 1];
    v2.z = shared_data[(thrdblck_id << 3) + 2];
    v2.w = shared_data[(thrdblck_id << 3) + 3];
    v4.x = shared_data[(thrdblck_id << 3) + 4];
    v4.y = shared_data[(thrdblck_id << 3) + 5];
    v4.z = shared_data[(thrdblck_id << 3) + 6];
    v4.w = shared_data[(thrdblck_id << 3) + 7];

    asm volatile ("add.cc.u32 %0, %1, %2;":"=r"(r[0]) : "r"(v1.x), "r"(v2.x));
    asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[1]) : "r"(v1.y), "r"(v2.y));
    asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[2]) : "r"(v1.z), "r"(v2.z));
    asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[3]) : "r"(v1.w), "r"(v2.w));
    asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[4]) : "r"(v3.x), "r"(v4.x));
    asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[5]) : "r"(v3.y), "r"(v4.y));
    asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[6]) : "r"(v3.z), "r"(v4.z));
    asm volatile ("addc.cc.u32 %0, %1, %2;":"=r"(r[7]) : "r"(v3.w), "r"(v4.w));
    asm volatile ("addc.u32 %0, 0, 0;": "=r"(r[8]));

#pragma unroll
        for (int k = 2; k < K_LEN; ++k)
        {
      shared_index[thrdblck_id] = ind[k];
      __syncthreads();

      shared_data[(hash_id << 3) + thread_id] = (hashes[(shared_index[hash_id] << 3) + thread_id]);
      shared_data[(hash_id << 3) + thread_id + 64] = (hashes[(shared_index[hash_id + 8] << 3) + thread_id]);
      shared_data[(hash_id << 3) + thread_id + 128] = (hashes[(shared_index[hash_id + 16] << 3) + thread_id]);
      shared_data[(hash_id << 3) + thread_id + 192] = (hashes[(shared_index[hash_id + 24] << 3) + thread_id]);
      shared_data[(hash_id << 3) + thread_id + 256] = (hashes[(shared_index[hash_id + 32] << 3) + thread_id]);
      shared_data[(hash_id << 3) + thread_id + 320] = (hashes[(shared_index[hash_id + 40] << 3) + thread_id]);
      shared_data[(hash_id << 3) + thread_id + 384] = (hashes[(shared_index[hash_id + 48] << 3) + thread_id]);
      shared_data[(hash_id << 3) + thread_id + 448] = (hashes[(shared_index[hash_id + 56] << 3) + thread_id]);
      __syncthreads();

      v1.x = shared_data[(thrdblck_id << 3) + 0];
      v1.y = shared_data[(thrdblck_id << 3) + 1];
      v1.z = shared_data[(thrdblck_id << 3) + 2];
      v1.w = shared_data[(thrdblck_id << 3) + 3];
      v2.x = shared_data[(thrdblck_id << 3) + 4];
      v2.y = shared_data[(thrdblck_id << 3) + 5];
      v2.z = shared_data[(thrdblck_id << 3) + 6];
      v2.w = shared_data[(thrdblck_id << 3) + 7];

      asm volatile ("add.cc.u32 %0, %0, %1;":"+r"(r[0]) : "r"(v1.x));
      asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[1]) : "r"(v1.y));
      asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[2]) : "r"(v1.z));
      asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[3]) : "r"(v1.w));
      asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[4]) : "r"(v2.x));
      asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[5]) : "r"(v2.y));
      asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[6]) : "r"(v2.z));
      asm volatile ("addc.cc.u32 %0, %0, %1;":"+r"(r[7]) : "r"(v2.w));
      asm volatile ("addc.u32 %0, %0, 0;": "+r"(r[8]));
        }

    for (j = 0; ctx->c < BUF_SIZE_8 && j < NUM_SIZE_8; ++j)
        {
      ctx->b[ctx->c++] = ((const uint8_t *)r)[NUM_SIZE_8 - j - 1];
        }



    B2B_IV(aux);
    B2B_IV(aux + 8);
    aux[0] = ivals[0];
    ((uint64_t *)(aux))[12] ^= 32;
    ((uint64_t *)(aux))[13] ^= 0;

    ((uint64_t *)(aux))[14] = ~((uint64_t *)(aux))[14];

    uint8_t *bb = (uint8_t *)(&(((uint64_t *)(aux))[16]));
    for (j = 0; j < NUM_SIZE_8; ++j)
        {
      bb[j] = ((const uint8_t *)r)[NUM_SIZE_8 - j - 1];
        }

    ((uint64_t *)(aux))[20] = 0;
    ((uint64_t *)(aux))[21] = 0;
    ((uint64_t *)(aux))[22] = 0;
    ((uint64_t *)(aux))[23] = 0;
    ((uint64_t *)(aux))[24] = 0;
    ((uint64_t *)(aux))[25] = 0;
    ((uint64_t *)(aux))[26] = 0;
    ((uint64_t *)(aux))[27] = 0;
    ((uint64_t *)(aux))[28] = 0;
    ((uint64_t *)(aux))[29] = 0;
    ((uint64_t *)(aux))[30] = 0;
    ((uint64_t *)(aux))[31] = 0;

    devB2B_MIX(aux, aux + 16);

    uint64_t hsh;
    uint32_t r_l[32];
#pragma unroll
    for (j = 0; j < NUM_SIZE_32; j += 2)
    {
      hsh = ivals[j >> 1];
      hsh ^= ((uint64_t *)(aux))[j >> 1] ^ ((uint64_t *)(aux))[8 + (j >> 1)];
      r_l[j] = ((uint32_t*)&hsh)[0];
      r_l[j+1] = ((uint32_t*)&hsh)[1];

    }

#pragma unroll 32
    for (j = 0; j < NUM_SIZE_8; j ++)
        {
      ((uint8_t *)r)[j] = ((uint8_t *)r_l)[NUM_SIZE_8 - j - 1];
        }

        j = ((uint64_t *)r)[3] < ((uint64_t *)bound_)[3]
            || ((uint64_t *)r)[3] == ((uint64_t *)bound_)[3] && (
                ((uint64_t *)r)[2] < ((uint64_t *)bound_)[2]
                || ((uint64_t *)r)[2] == ((uint64_t *)bound_)[2] && (
                    ((uint64_t *)r)[1] < ((uint64_t *)bound_)[1]
                    || ((uint64_t *)r)[1] == ((uint64_t *)bound_)[1]
                    && ((uint64_t *)r)[0] < ((uint64_t *)bound_)[0]
                )
            );


    if (j)
            {


                uint32_t id = atomicInc(count, MAX_SOLS);
      valid[id] = tid + 1;
           }


    }

    return;


}

